#include "hip/hip_runtime.h"
#include <iostream>

#include <cmdparser.h>
#include <array.h>
#include <matrix.h>
#include <util.h>
#include <utility.h>
#include <profile.h>
#include <trainable_dtw.h>

//#include <cdtw.h>

using namespace DtwUtil;
using namespace std;


vector<size_t> theta;
Model model;

//typedef Matrix2D<double> mat;

void dumpMfccAsKaldiArk(const Array<string>& lists);
void normalize(mat& m, int type = 1);
// double cdtw(DtwParm& q_parm, DtwParm& d_parm);
double cdtw(const string& f1, const string& f2);
void chooseLargestGranularity(const string& path, Array<string>& lists);
enum DTW_TYPE { FIXDTW, FFDTW, SCDTW, CDTW };
DTW_TYPE getDtwType(const string& typeStr);

template <typename T>
double other_dtw(DtwParm& q_parm, DtwParm& d_parm) {
  vector<float> hypo_score;
  vector<pair<int, int> > hypo_bound;

  FrameDtwRunner::nsnippet_ = 10;
  T dtwRunner = T(DtwUtil::euclinorm);
  dtwRunner.InitDtw(&hypo_score, &hypo_bound, NULL, &q_parm, &d_parm, NULL, NULL);
  dtwRunner.DTW();

  float max = hypo_score[0];
  foreach (i, hypo_bound) {
    if (hypo_score[i] > max)
      max = hypo_score[i];
  }
  return (double) max;
}

template <typename T>
std::pair<T, T> getMinMax(const Matrix2D<T>& m) {
  double min = m[0][0];
  double max = m[0][0];

  size_t rows = m.getRows();
  size_t cols = m.getCols();

  for (size_t i=0; i<rows; ++i) {
    for (size_t j=0; j<cols; ++j) {
      T e = m[i][j];
      min = (e < min) ? e : min;
      max = (e > max) ? e : max;
    }
  }

  return std::pair<T, T>(min, max);
}

int main (int argc, char* argv[]) {

  CmdParser cmdParser(argc, argv);
  cmdParser
    .add("-d", "directory containing mfcc files for a certain query")
    .add("-o", "output filename for the acoustic similarity matrix")
    .add("--list", "corresponding list of mfcc files")
    .add("--dtw-type", "Choose the type of Dynamic Time Warping: \n"
			  "fixdtw:\t FixFrameDtwRunner. head-to-head, tail-to-tail\n"
			  "ffdtw:\t FreeFrameDtwRunner. no head-to-head, tail-to-tail constraint\n"
			  "scdtw:\t SlopeConDtwRunner. Slope-conditioned DTW\n"
			  "cdtw:\t CumulativeDtwRunner. Cumulative DTW, considering all paths from head-to-tail.")
    .add("--normalize", "Whether to normalize the acoustic similarity to [0, 1]", false, "true");

  cmdParser
    .addGroup("Distance options")
    .add("--theta", "specify the file containing the diagnol term of Mahalanobis distance (dim=39)", false)
    .add("--eta", "Specify the coefficient in the smoothing minimum", false, "-4");

  if(!cmdParser.isOptionLegal())
    cmdParser.showUsageAndExit();

  Profile profile;
  profile.tic();

  model.load("data/dtwdnn.model/");
  // =====================================================
  string path = cmdParser.find("-d") + "/";
  string mat_filename = cmdParser.find("-o");
  string list_filename = cmdParser.find("--list");
  bool normalization = cmdParser.find("--normalize") == "false" ? false : true;
  string theta_filename = cmdParser.find("--theta");
  SMIN::eta = str2double(cmdParser.find("--eta"));

  Bhattacharyya::setDiagFromFile(theta_filename);

  DTW_TYPE type = getDtwType(cmdParser.find("--dtw-type"));

  Array<string> lists(list_filename);
  chooseLargestGranularity(path, lists);

  // dumpMfccAsKaldiArk(lists);
  // return 0;

  int nSegment = lists.size();

  vector<DtwParm> parms;
  foreach (i, lists)
    parms.push_back(DtwParm(lists[i]));

  mat scores(nSegment, nSegment);

  range (i, nSegment) {

    range (j, nSegment) {
      if (j > i) break;
      // cout << "(i, j) = (" << i << ", " << j << ")" << endl;

      double score = 0;
      switch (type) {
	case CDTW:
	  score = cdtw(lists[i], lists[j]);
	  // FIXME DtwParm seemed to have illed copy constructor !!!
	  // score = dtwdnn::dtw(lists[i], lists[j]);
	  break;
	/*case FIXDTW:
	  score = other_dtw<FixFrameDtwRunner>(parms[i], parms[j]);
	  break;
	case SCDTW:
	  score = other_dtw<SlopeConDtwRunner>(parms[i], parms[j]);
	  break;
	case FFDTW:
	default:
	  score = other_dtw<FreeFrameDtwRunner>(parms[i], parms[j]);
	  break;*/
      }

      scores[i][j] = scores[j][i] = score;
    }
  }

  normalize(scores, 1);
  scores.saveas(mat_filename);

  cout << endl;
  profile.toc();

  return 0;
}

void dumpMfccAsKaldiArk(const Array<string>& lists) {

  foreach (i, lists) {
    cout << lists[i] << "  [" << endl;

    DtwParm p(lists[i]);
    size_t feat_dim = p.Feat().LF();
    size_t totalTime = p.Feat().LT();
    for (int t=0; t<totalTime; ++t) {
      cout << "  ";

      for (int d=0; d<feat_dim; ++d)
	cout << p.Feat()[t][d] << " ";

      if (t != totalTime - 1)
	cout << endl;
      else
	cout << "]" << endl;
    }
  }
}

double cdtw(const string& f1, const string& f2) {
  vector<float> hypo_score;
  vector<pair<int, int> > hypo_bound;

  FrameDtwRunner::nsnippet_ = 10;

  DtwParm q_parm(f1);
  DtwParm d_parm(f2);
  CumulativeDtwRunner dtwRunner = CumulativeDtwRunner(Bhattacharyya::fn);
  dtwRunner.InitDtw(&hypo_score, &hypo_bound, NULL, &q_parm, &d_parm, NULL, NULL);
  dtwRunner.DTW(true);

  double cScoreInLog = dtwRunner.getCumulativeScore();
  return -cScoreInLog;
}

DTW_TYPE getDtwType(const string& typeStr) {
  if (typeStr == "fixdtw")
    return FIXDTW;
  else if (typeStr == "scdtw")
    return SCDTW;
  else if (typeStr == "cdtw")
    return CDTW;
  else
    return FFDTW;
}

void chooseLargestGranularity(const string& path, Array<string>& lists) {
  // Choose Highest number. (i.e. largest granularity)
  // Granularity: word > character > syllable > phone
  foreach (i, lists) {
    for (int j=1; j<50; ++j) {
      string filename = path + lists[i] + "_" + int2str(j) + ".gp";
      if (exists(filename)) {
	lists[i] = filename;
	break;
      }
    }
  }
}

void normalize(mat& m, int type) {
  std::pair<float, float> minmax = getMinMax(m);
  float min = minmax.first;
  float max = minmax.second;

  switch (type) {
    // [min, max] ==> [min - max, 0] ==> [-1, 0] ==> [0, 1]
    //           shift              scale       shift
    case 1:
      m -= max;
      m /= (max - min);
      m += 1;
      break;

    // [min, max] ==> [min - max, 0] ==> [0, 1]
    //          shift                exp
    case 2:
      m -= max;
      range(i, m.getRows())
	range(j, m.getCols())
	  m[i][j] = exp(m[i][j]);
      
      break;
  }
}


// double scdtw(DtwParm& q_parm, DtwParm& d_parm);
// double ffdtw(DtwParm& q_parm, DtwParm& d_parm);
// double fixdtw(DtwParm& q_parm, DtwParm& d_parm);
/*double ffdtw(DtwParm& q_parm, DtwParm& d_parm) {
  vector<float> hypo_score;
  vector<pair<int, int> > hypo_bound;

  FrameDtwRunner::nsnippet_ = 10;
  FreeFrameDtwRunner dtwRunner = FreeFrameDtwRunner(DtwUtil::euclinorm);
  dtwRunner.InitDtw(&hypo_score, &hypo_bound, NULL, &q_parm, &d_parm, NULL, NULL);
  dtwRunner.DTW();

  float max = numeric_limits<float>::lowest();
  foreach (i, hypo_bound) {
    if (hypo_score[i] > max)
      max = hypo_score[i];
  }
  return (double) max;
}

double fixdtw(DtwParm& q_parm, DtwParm& d_parm) {
  vector<float> hypo_score;
  vector<pair<int, int> > hypo_bound;

  FrameDtwRunner::nsnippet_ = 10;
  FixFrameDtwRunner dtwRunner = FixFrameDtwRunner(DtwUtil::euclinorm);
  dtwRunner.InitDtw(&hypo_score, &hypo_bound, NULL, &q_parm, &d_parm, NULL, NULL);
  dtwRunner.DTW();

  float max = numeric_limits<float>::lowest();
  foreach (i, hypo_bound) {
    if (hypo_score[i] > max)
      max = hypo_score[i];
  }
  return (double) max;
}

double scdtw(DtwParm& q_parm, DtwParm& d_parm) {
  vector<float> hypo_score;
  vector<pair<int, int> > hypo_bound;

  FrameDtwRunner::nsnippet_ = 10;
  SlopeConDtwRunner dtwRunner = SlopeConDtwRunner(DtwUtil::euclinorm);
  dtwRunner.InitDtw(&hypo_score, &hypo_bound, NULL, &q_parm, &d_parm, NULL, NULL);
  dtwRunner.DTW();

  float max = numeric_limits<float>::lowest();
  foreach (i, hypo_bound) {
    if (hypo_score[i] > max)
      max = hypo_score[i];
  }
  return (double) max;
}
*/
