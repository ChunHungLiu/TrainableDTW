#include <dnn.h>
#include <utility.h>

vec loadvector(string filename) {
  Array<float> arr(filename);
  vec v(arr.size());
  foreach (i, arr)
    v[i] = arr[i];
  return v;
}

DNN::DNN() {}

DNN::DNN(const vector<size_t>& dims): _dims(dims) {
  _weights.resize(_dims.size() - 1);

  foreach (i, _weights) {
    size_t M = _dims[i] + 1;
    size_t N = _dims[i + 1];
    _weights[i].resize(M, N);
  }

  randInit();
}

DNN::DNN(const DNN& source): _dims(source._dims), _weights(source._weights) {
}

DNN& DNN::operator = (DNN rhs) {
  swap(*this, rhs);
  return *this;
}

size_t DNN::getNLayer() const {
  return _dims.size(); 
}

size_t DNN::getDepth() const {
  return _dims.size() - 2;
}

void DNN::print() const {
  foreach (i, _weights)
    _weights[i].print(5);
}

void DNN::getEmptyGradient(vector<mat>& g) const {
  g.resize(_weights.size());
  foreach (i, _weights) {
    int m = _weights[i].getRows();
    int n = _weights[i].getCols();
    g[i].resize(m, n);
  }
}

vector<mat>& DNN::getWeights() { return _weights; }
const vector<mat>& DNN::getWeights() const { return _weights; }
vector<size_t>& DNN::getDims() { return _dims; }
const vector<size_t>& DNN::getDims() const { return _dims; }

void DNN::randInit() {
  foreach (i, _weights)
    ext::rand(_weights[i]);
}

// ========================
// ===== Feed Forward =====
// ========================
void DNN::feedForward(const vec& x, vector<vec>* hidden_output) {

  vector<vec>& O = *hidden_output;
  //O.resize(_dims.size());
  assert(O.size() == _dims.size());

  // Init with one extra element, which is bias
  O[0].resize(x.size() + 1);
  std::copy(x.begin(), x.end(), O[0].begin());

  for (size_t i=1; i<O.size() - 1; ++i)
    O[i] = ext::b_sigmoid(O[i-1] * _weights[i-1]);

  size_t end = O.size() - 1;
  O[end] = ext::sigmoid(O[end - 1] * _weights[end - 1]);
}

// ============================
// ===== Back Propagation =====
// ============================
vec DNN::backPropagate(const vec& x, vector<vec>* O, vector<mat>* gradient) {

  assert(gradient->size() == _weights.size());

  vec p(x);
  reverse_foreach (i, _weights) {
    (*gradient)[i] = (*O)[i] * p;
    p = (*O)[i] & ( (float) 1.0 - (*O)[i] ) & (_weights[i] * p);

    // Remove bias
    p.pop_back();
  }

  return p;
}

void swap(DNN& lhs, DNN& rhs) {
  using std::swap;
  swap(lhs._dims   , rhs._dims   );
  swap(lhs._weights, rhs._weights);
}

void swap(HIDDEN_OUTPUT& lhs, HIDDEN_OUTPUT& rhs) {
  using std::swap;
  swap(lhs.hox, rhs.hox);
  swap(lhs.hoy, rhs.hoy);
  swap(lhs.hoz, rhs.hoz);
  swap(lhs.hod, rhs.hod);
}

void swap(GRADIENT& lhs, GRADIENT& rhs) {
  using std::swap;
  swap(lhs.grad1, rhs.grad1);
  swap(lhs.grad2, rhs.grad2);
  swap(lhs.grad3, rhs.grad3);
  swap(lhs.grad4, rhs.grad4);
}
// ===============================
// ===== Class DTW-DNN Model =====
// ===============================
Model::Model() {}

Model::Model(const vector<size_t>& pp_dim, const vector<size_t>& dtw_dim): _lr(-0.0001), _pp(pp_dim), _dtw(dtw_dim) {
  _w = ext::rand<float>(_dtw.getDims()[0]);
  this->initHiddenOutputAndGradient();
}

Model::Model(const Model& source): gradient(source.gradient), hidden_output(source.hidden_output), _lr(source._lr), _pp(source._pp), _w(source._w), _dtw(source._dtw) {}

Model& Model::operator = (Model rhs) {
  swap(*this, rhs);
  return *this;
}

void Model::initHiddenOutputAndGradient() {

  hidden_output.hox.resize(_pp.getNLayer());
  hidden_output.hoy.resize(_pp.getNLayer());
  hidden_output.hod.resize(_dtw.getNLayer());

  gradient.grad1.resize(_pp.getWeights().size());
  gradient.grad2.resize(_pp.getWeights().size());
  gradient.grad4.resize(_dtw.getWeights().size());
}

float Model::evaluate(const float* x, const float* y) {
  int length = _pp.getDims()[0];
  return this->evaluate(vec(x, x+length), vec(y, y+length));
}

float Model::evaluate(const vec& x, const vec& y) {

  HIDDEN_OUTPUT_ALIASING(hidden_output, Ox, Oy, Om, Od);

  _pp.feedForward(x, &Ox);
  _pp.feedForward(y, &Oy);

  Ox.back() = ext::softmax(Ox.back());
  Oy.back() = ext::softmax(Oy.back());

  Om = Ox.back() & Oy.back() & _w;

  _dtw.feedForward(Om, &Od);

  float d = Od[Od.size() - 1][0];
  return d;
}

void Model::train(const vec& x, const vec& y) {
  this->evaluate(x, y);
  this->calcGradient(x, y);
  this->updateParameters(this->gradient);
}

void Model::calcGradient(const float* x, const float* y) {
  int length = _pp.getDims()[0];
  this->calcGradient(vec(x, x + length), vec(y, y+length));
}

void Model::calcGradient(const vec& x, const vec& y) {

  HIDDEN_OUTPUT_ALIASING(hidden_output, Ox, Oy, Om, Od);
  GRADIENT_ALIASING(gradient, ppg1, ppg2, middle_gradient, dtw_gradient);
  // ==============================================
  vec& final_output = Od.back();
  vec p = _dtw.backPropagate(final_output, &Od, &dtw_gradient);

  // ==============================================
  middle_gradient = Om & p;

  vec px = p & Oy.back() & _w;
  vec py = p & Ox.back() & _w;

  px = (px - ext::sum(px & Ox.back()) ) & Ox.back();
  py = (py - ext::sum(py & Oy.back()) ) & Oy.back();

  // ==============================================
  _pp.backPropagate(px, &Ox, &ppg1);
  _pp.backPropagate(py, &Oy, &ppg2);
}

void Model::updateParameters(GRADIENT& g) {
  GRADIENT_ALIASING(g, ppg1, ppg2, mg, dtwg);

  vector<mat>& ppw = _pp.getWeights();
  foreach (i, ppw)
    ppw[i] += _lr * (ppg1[i] + ppg2[i]); 

  this->_w += _lr * mg;

  vector<mat>& dtww = _dtw.getWeights();
  foreach (i, dtww)
    dtww[i] += _lr * dtwg[i];
}

void Model::setLearningRate(float learning_rate) {
  _lr = learning_rate;
}

HIDDEN_OUTPUT& Model::getHiddenOutput() {
  return hidden_output;
}

GRADIENT& Model::getGradient() {
  return gradient;
}

void Model::getEmptyGradient(GRADIENT& g) {
  GRADIENT_ALIASING(g, g1, g2, g3, g4);

  _pp.getEmptyGradient(g1);
  _pp.getEmptyGradient(g2);

  g3.resize(_dtw.getDims()[0]);

  _dtw.getEmptyGradient(g4);
}

void Model::load(string folder) {
  folder += "/";
  
  vector<mat>& ppw = _pp.getWeights();
  foreach (i, ppw)
    ppw[i] = mat(folder + "pp.w." + int2str(i));

  ext::load<float>(this->_w, folder + "m.w");

  vector<mat>& dtww = _dtw.getWeights();
  foreach (i, dtww)
    dtww[i] = mat(folder + "dtw.w." + int2str(i));
}

void Model::save(string folder) const {

  folder += "/";
  
  const vector<mat>& ppw = _pp.getWeights();
  foreach (i, ppw)
    ppw[i].saveas(folder + "pp.w." + int2str(i));

  ext::save(this->_w, folder + "m.w");

  const vector<mat>& dtww = _dtw.getWeights();
  foreach (i, dtww)
    dtww[i].saveas(folder + "dtw.w." + int2str(i));
}

void Model::print() const {
  _pp.print();
  ::print(_w);
  _dtw.print();
}

void swap(Model& lhs, Model& rhs) {
  using std::swap;
  swap(lhs.hidden_output, rhs.hidden_output);
  swap(lhs.gradient, rhs.gradient);
  swap(lhs._lr , rhs._lr );
  swap(lhs._pp , rhs._pp );
  swap(lhs._w  , rhs._w  );
  swap(lhs._dtw, rhs._dtw);
}


GRADIENT& operator += (GRADIENT& g1, GRADIENT& g2) {
  GRADIENT_ALIASING(g1, g1_1, g1_2, g1_3, g1_4);
  GRADIENT_ALIASING(g2, g2_1, g2_2, g2_3, g2_4);

  foreach (i, g1_1)
    g1_1[i] += g2_1[i];

  foreach (i, g1_2)
    g1_2[i] += g2_2[i];

  g1_3 += g2_3; 

  foreach (i, g1_4)
    g1_4[i] += g2_4[i];

  return g1;
}

GRADIENT& operator -= (GRADIENT& g1, GRADIENT& g2) {
  GRADIENT_ALIASING(g1, g1_1, g1_2, g1_3, g1_4);
  GRADIENT_ALIASING(g2, g2_1, g2_2, g2_3, g2_4);

  foreach (i, g1_1) g1_1[i] -= g2_1[i];
  foreach (i, g1_2) g1_2[i] -= g2_2[i];
  g1_3 -= g2_3; 
  foreach (i, g1_4) g1_4[i] -= g2_4[i];

  return g1;
}

GRADIENT& operator *= (GRADIENT& g, float c) {
  GRADIENT_ALIASING(g, g1, g2, g3, g4);

  foreach (i, g1) g1[i] *= c;
  foreach (i, g2) g2[i] *= c;
  g3 *= c;
  foreach (i, g4) g4[i] *= c;

  return g;
}

GRADIENT& operator /= (GRADIENT& g, float c) {
  return (g *= (float) 1.0 / c);
}

GRADIENT operator + (GRADIENT g1, GRADIENT& g2) { return (g1 += g2); }
GRADIENT operator - (GRADIENT g1, GRADIENT& g2) { return (g1 -= g2); }
GRADIENT operator * (GRADIENT g, float c) { return (g *= c); }
GRADIENT operator * (float c, GRADIENT g) { return (g *= c); }
GRADIENT operator / (GRADIENT g, float c) { return (g /= c); }

void print(GRADIENT& g) {
  GRADIENT_ALIASING(g, g1, g2, g3, g4);
  
  foreach (i, g1)
    g1[i].print();

  foreach (i, g2)
    g2[i].print();

  cout << endl;
  print(g3);

  foreach (i, g4)
    g4[i].print();
}
