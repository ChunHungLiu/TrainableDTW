#include "hip/hip_runtime.h"
#include <fast_dtw.h>
#define __pow__(x) ((x)*(x))

__device__ float 
euclidean(const float* x, const float* y, size_t dim) {
  float d = 0;
  for (size_t i=0; i<dim; ++i)
    d += __pow__(x[i] - y[i]);
  return sqrt(d);
}

__global__ void pairWiseKernel(const float* f1, const float* f2, size_t rows, size_t cols, size_t dim, float* pdist) {

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if(x < 0 || x > rows-1 || y < 0 || y > cols-1)
    return;

  int index = x * cols + y; 
  pdist[index] = euclidean(f1 + x*dim, f2 + y * dim, dim);
}

float pair_distance(const float* f1, const float* f2, size_t rows, size_t cols, size_t dim, float eta, float* pdist, distance_fn& d) {
  for (int x = 0; x < rows; ++x)
    for (int y = 0; y < cols; ++y)
      pdist[x * cols + y] = d(f1 + x * dim, f2 + y * dim, dim);
}

float pair_distance_in_gpu(const float* f1, const float* f2, size_t w, size_t h, size_t dim, float eta, float* pdist, hipStream_t& stream) {
  const int BLOCK_SIZE = 8;
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(w / BLOCK_SIZE, h / BLOCK_SIZE);
  if(w % BLOCK_SIZE > 0) ++grid.x;
  if(h % BLOCK_SIZE > 0) ++grid.y;

  pairWiseKernel<<<grid, threads, 0, stream>>>(f1, f2, w, h, dim, pdist);
}

float pair_distance_in_gpu(const float* f1, const float* f2, size_t w, size_t h, size_t dim, float eta, float* pdist) {
  const int BLOCK_SIZE = 64;
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(w / BLOCK_SIZE, h / BLOCK_SIZE);
  if(w % BLOCK_SIZE > 0) ++grid.x;
  if(h % BLOCK_SIZE > 0) ++grid.y;

  pairWiseKernel<<<grid, threads>>>(f1, f2, w, h, dim, pdist);
}


void callback(hipStream_t stream, hipError_t status, void* userData) {

  float* pdist = ((P_DIST*) userData)->pdist;
  int w = ((P_DIST*) userData)->w;
  int h = ((P_DIST*) userData)->h;
  int dim = ((P_DIST*) userData)->dim;
  float* d = ((P_DIST*) userData)->d;

  *d = fast_dtw(pdist, w, h, dim, -4, NULL, NULL);

  StreamManager::getInstance().pop_front();
}

float* computePairwiseDTW(const float* data, const unsigned int* offset, int N, int dim, distance_fn& fn, float eta) {

  size_t MAX_LENGTH = 0;
  range (i, N) {
    unsigned int length = (offset[i+1] - offset[i]) / dim;
    if ( length > MAX_LENGTH)
	MAX_LENGTH = length;
  }

  float* alpha = new float[MAX_LENGTH * MAX_LENGTH];
  float* pdist = new float[MAX_LENGTH * MAX_LENGTH];

  float* scores = new float[N * N];

  for (int i=0; i<N; ++i) {

    scores[i * N + i] = 0;
    for (int j=0; j<i; ++j) {
      size_t length1 = (offset[i + 1] - offset[i]) / dim;
      size_t length2 = (offset[j + 1] - offset[j]) / dim;

      const float *f1 = data + offset[i];
      const float *f2 = data + offset[j];

      pair_distance(f1, f2, length1, length2, dim, eta, pdist, fn);
      float s = fast_dtw(pdist, length1, length2, dim, eta, alpha);
      scores[i * N + j] = scores[j * N + i] = s;
    }
  }

  delete [] alpha;
  delete [] pdist;

  return scores;
}

float* computePairwiseDTW_in_gpu(const float* data, const unsigned int* offset, int N, int dim) {

  size_t MAX_LENGTH = findMaxLength(offset, N, dim);

  size_t size = (size_t) offset[N] * sizeof(float);
  size_t offsetSize = (N + 1) * sizeof(unsigned int);
  size_t MAX_TABLE_SIZE = MAX_LENGTH * MAX_LENGTH * sizeof(float);

  float* d_scores;
  float* d_data;
  float* d_pdist;
  float* d_alpha;
  unsigned int* d_offset;

  CCE(hipMalloc((void**) &d_scores, N * N * sizeof(float)));
  CCE(hipMalloc((void**) &d_data, size));
  CCE(hipMalloc((void**) &d_offset, offsetSize));
  CCE(hipMalloc((void**) &d_alpha, MAX_TABLE_SIZE));

  CCE(hipMemset(d_scores, 0, N * N * sizeof(float)));
  CCE(hipMemcpy(d_data, data, size, hipMemcpyHostToDevice));
  CCE(hipMemcpy(d_offset, offset, offsetSize, hipMemcpyHostToDevice));
  // No need to copy h_pdist and h_alpha since they're only buffers
  CCE(hipDeviceSynchronize());

  float* scores = new float[N * N];
  for (int i=0; i<N*N; ++i)
    scores[i] = 0;

  size_t nStream = StreamManager::getInstance().size();
  float* pdist = new float[nStream * MAX_TABLE_SIZE];
  CCE(hipMalloc((void**) &d_pdist, nStream * MAX_TABLE_SIZE));

  // ===== Begin of Dynamic Time Warping =====
  for (int i=0; i<N; ++i) {
    scores[i*N + i] = 0;
    for (int j=0; j<i; ++j) {

      size_t w = (offset[i + 1] - offset[i]) / dim,
	     h = (offset[j + 1] - offset[j]) / dim;

      float *d_f1 = d_data + offset[i],
	    *d_f2 = d_data + offset[j];

      while( ! StreamManager::getInstance().push_back(d_f1, d_f2, w, h, dim, MAX_TABLE_SIZE, d_pdist, pdist, &scores[i*N + j]) );

      /*pair_distance_in_gpu(d_f1, d_f2, w, h, dim, -4, d_pdist);
      CCE(hipDeviceSynchronize());
      CCE(hipMemcpy(pdist, d_pdist, MAX_TABLE_SIZE, hipMemcpyDeviceToHost));
      float dist = fast_dtw(data + offset[i], data + offset[j], w, h, dim, -4, pdist, NULL, NULL);
      scores[i * N + j] = scores[j * N + i] = dist;
      CCE(hipDeviceSynchronize()); */
    }
  }
  CCE(hipDeviceSynchronize());

  range (i, N)
    range (j, i)
      scores[j * N + i] = scores[i * N + j];

  delete [] pdist;

  // ===== End of Dynamic Time Warping =====

  CCE(hipFree(d_scores));
  CCE(hipFree(d_data));
  CCE(hipFree(d_offset));
  CCE(hipFree(d_pdist));
  CCE(hipFree(d_alpha));

  CCE(hipDeviceSynchronize());

  return scores;
}

inline float addlog(float x, float y) {
  const float MAX_DIFF = -708;

  if (x < y)
    std::swap(x, y);

  float diff = y - x;
  if ( diff < MAX_DIFF )
    return x;

  return x + log(1.0 + exp(diff));
}

inline float smin(float x, float y, float z, float eta) {
  return addlog(addlog(eta * x, eta * y), eta * z) / eta;
}

size_t findMaxLength(const unsigned int* offset, int N, int dim) {
  size_t MAX_LENGTH = 0;
  range (i, N) {
    unsigned int length = (offset[i+1] - offset[i]) / dim;
    if ( length > MAX_LENGTH)
	MAX_LENGTH = length;
  }
  return MAX_LENGTH;
}

float** malloc2D(size_t m, size_t n) {
  float** p = new float*[m];
  range (i, m)
    p[i] = new float[n];
  return p;
}

void free2D(float** p, size_t m) {
  assert(p != NULL);

  range (i, m)
    delete p[i];
  delete [] p;
}

float fast_dtw(float* pdist, size_t rows, size_t cols, size_t dim, float eta, float* alpha, float* beta) {
  
  float distance = 0;

  bool isAlphaNull = (alpha == NULL);

  if (isAlphaNull)
    alpha = new float[rows * cols];

  // ===== Begin of Main =====
  // x == y == 0 
  alpha[0] = pdist[0];

  // y == 0
  for (int x = 1; x < rows; ++x)
    alpha[x * cols] = alpha[(x-1) * cols] + pdist[x * cols];

  // x == 0
  for (int y = 1; y < cols; ++y)
    alpha[y] = alpha[y-1] + pdist[y];

  // interior points
  for (int x = 1; x < rows; ++x) {
    for (int y = 1; y < cols; ++y) {
      alpha[x * cols + y] = (float) smin(alpha[(x-1) * cols + y], alpha[x * cols + y-1], alpha[(x-1) * cols + y-1], eta) + pdist[x * cols + y];
    }
  }

  distance = alpha[rows * cols - 1];
  // ====== End of Main ======

  if (beta != NULL) {
    beta[rows * cols - 1] = 0;
    int x, y;
    y = cols - 1;
    for (x = rows - 2; x >= 0; --x)
      beta[x * cols + y] = beta[(x+1) * cols + y] + pdist[(x+1) * cols + y];

    x = rows - 1;
    for (y = cols - 2; y >= 0; --y)
      beta[x * cols + y] = beta[x * cols + (y+1)] + pdist[x * cols + (y+1)];

    for (x = rows - 2; x >= 0; --x) {
      for (y = cols - 2; y >= 0; --y) {
	int p1 =  x    * cols + y + 1,
	    p2 = (x+1) * cols + y    ,
	    p3 = (x+1) * cols + y + 1;

	float s1 = beta[p1] + pdist[p1],
	      s2 = beta[p2] + pdist[p2],
	      s3 = beta[p3] + pdist[p3];

	beta[x * cols + y] = smin(s1, s2, s3, eta);
      }
    }
  }

  if (isAlphaNull) delete [] alpha;

  return distance;
}

bool StreamManager::pop_front() {
  --_counter;
  _userData.pop();
}

bool StreamManager::push_back(const float* f1, const float* f2, int w, int h, int dim, int MAX_TABLE_SIZE, float* d_pdist, float* pdist, float* d) {

  if (_counter + 1 >= _nStream)
    return false;

  hipStream_t& s = this->_stream[_counter];

  size_t offset = _counter * MAX_TABLE_SIZE;
  pair_distance_in_gpu(f1, f2, w, h, dim, -4, d_pdist + offset, s);
  CCE(hipMemcpyAsync(pdist + offset, d_pdist + offset, MAX_TABLE_SIZE, hipMemcpyDeviceToHost, s));

  _userData.push(P_DIST(pdist + offset, w, h, dim, d));
  hipStreamAddCallback(s, ::callback, &(_userData.back()), 0);

  _counter++;

  return true;
}

size_t StreamManager::size() { return _nStream; }

StreamManager& StreamManager::getInstance() {
  static StreamManager instance(128);
  return instance;
}

StreamManager::StreamManager(int nStream):_nStream(nStream), _counter(0) {
  _stream = new hipStream_t[_nStream];
  range (i, _nStream)
    CCE(hipStreamCreate(&_stream[i]));
}

StreamManager::~StreamManager() {
  range (i, _nStream)
    CCE(hipStreamDestroy(_stream[i]));
}

/*extern "C" __global__
void dtwKernel(float* distance, const float* f1, const float* f2, size_t w, size_t h, size_t dim, float eta, float* pdist, float* alpha, float* beta) {

  // y == x == 0 
  alpha[0] = pdist[0];

  // x == 0
  for (int y = 1; y < h; ++y)
    alpha[y * w] = alpha[(y-1) * w] + pdist[y * w];

  // y == 0
  for (int x = 1; x < w; ++x)
    alpha[x] = alpha[x-1] + pdist[x];

  float temp;
  // interior points
  for (int y = 1; y < h; ++y) {
    for (int x = 1; x < w; ++x) {
      temp = fmin(alpha[(y-1) * w + x], alpha[y * w + x-1]);
      alpha[y * w + x] = fmin(temp, alpha[(y-1) * w + x-1]) + pdist[y * w + x];
      //alpha[y * w + x] = (float) smin(alpha[(y-1) * w + x], alpha[y * w + x-1], alpha[(y-1) * w + x-1], eta) + pdist[y * w + x];
    }
  }

  *distance = alpha[h * w - 1];
}*/

/*float fast_dtw(const float* const* f1, const float* const* f2, size_t rows, size_t cols, size_t dim, float eta, float** pdist, float** alpha, float** beta) {

  float distance = 0;

  bool isAlphaNull = (alpha == NULL),
       isBetaNull  = (beta  == NULL),
       isPdistNull = (pdist == NULL);

  if (isAlphaNull)
    alpha = malloc2D(rows, cols);

  if (isPdistNull)
    pdist = malloc2D(rows, cols);
  // ===== Pre-calculate Pair-Wise Distance "pdist" =====
  for (int x = 0; x < rows; ++x)
    for (int y = 0; y < cols; ++y)
      pdist[x][y] = euclidean(f1[x], f2[y], dim);

  // ===== Begin of Main =====
  // x == y == 0 
  alpha[0][0] = pdist[0][0];

  // y == 0
  for (int x = 1; x < rows; ++x)
    alpha[x][0] = alpha[x-1][0] + pdist[x][0];

  // x == 0
  for (int y = 1; y < cols; ++y)
    alpha[0][y] = alpha[0][y-1] + pdist[0][y];

  // interior points
  for (int x = 1; x < rows; ++x)
    for (int y = 1; y < cols; ++y)
      alpha[x][y] = (float) smin(alpha[x-1][y], alpha[x][y-1], alpha[x-1][y-1], eta) + pdist[x][y];

  distance = alpha[rows - 1][cols - 1];
  // ====== End of Main ======
  
  if (beta != NULL) {
    // TODO
  }
  
  if (isAlphaNull)
    free2D(alpha, rows);

  if (isPdistNull)
    free2D(pdist, rows);

  return distance;
}

void loadKaldiArchive(string filename, vector<float**> &data, vector<size_t> &lengths, int &N, int &dim) {

  vector<FeatureSeq> featureSeqs;

  FILE* fptr = fopen(filename.c_str(), "r");
  vulcan::VulcanUtterance vUtterance;
  while (vUtterance.LoadKaldi(fptr))
    featureSeqs.push_back(vUtterance._feature);
  fclose(fptr);

  dim = featureSeqs[0][0].size();
  N = featureSeqs.size();
  lengths.resize(N);
  data.resize(N);

  range (i, N) {
    size_t length = featureSeqs[i].size();
    lengths[i] = length;

    data[i] = new float*[length];
    range (j, length) {
      data[i][j] = new float[dim];

      range(k, dim)
	data[i][j][k] = featureSeqs[i][j]._data->data[k];
    }
  }
}

void computePairwiseDTW(string filename, float** &scores, int& N) {

  int dim;
  vector<float**> data;
  vector<size_t> lengths;
  loadKaldiArchive(filename, data, lengths, N, dim);

  const size_t MAX_ROWS = 256;
  const size_t MAX_COLS = 256;

  float** alpha = malloc2D(MAX_ROWS, MAX_COLS);
  float** pdist = malloc2D(MAX_ROWS, MAX_COLS);

  scores = malloc2D(N, N);

  range (i, N) {
    range (j, i) {
      size_t rows = lengths[i];
      size_t cols = lengths[j];
      float s = fast_dtw(data[i], data[j], rows, cols, dim, -4, alpha);
      scores[i][j] = scores[j][i] = s;
    }
  }

  free2D(alpha, MAX_ROWS);
  free2D(pdist, MAX_ROWS);

  range (i, N)
    free2D(data[i], lengths[i]);
}*/

